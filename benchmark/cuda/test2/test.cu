#include "hip/hip_runtime.h"
#include <unistd.h>
#include <string.h>

#include <iostream>
#include <string>
#include <thread>
#include <mutex>

#include "../../../source/utils/common/time.h"

using namespace std;
using namespace balance::util;

mutex _lock;

__global__
void saxpy(int n, float a, float *x, float *y)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n) y[i] = a * x[i] + y[i];
}

void func1(void) {
	int N = 1 << 25;
	float *x, *y, *d_x, *d_y;
	x = (float*)malloc(N * sizeof(float));
	y = (float*)malloc(N * sizeof(float));

	hipMalloc(&d_x, N * sizeof(float)); 
	hipMalloc(&d_y, N * sizeof(float));

	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

	_lock.lock();
	cout << "func1 locks" << endl;
	usleep(2000);

    double time_1 = time_now();
    for (int i = 0; i < 1000; ++i) {
        saxpy<<<(N + 255) / 256, 256>>>(N, 2.0f, d_x, d_y);
    }
    // hipDeviceSynchronize();
    double time_2 = time_now();
    double latency = time_2 - time_1;
    cout << "Latency for computation: " << fixed << latency << endl;

	hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

	float maxError = 0.0f;
	for (int i = 0; i < N; i++)
		maxError = max(maxError, abs(y[i] - 2.0f * 1001));
	cout << "Max error: " << fixed << maxError << endl;

	hipFree(d_x);
	hipFree(d_y);
	free(x);
	free(y);
}

void func2(void) {
	int N = 1 << 25;
	float *x, *y, *d_x, *d_y;
	x = (float*)malloc(N * sizeof(float));
	y = (float*)malloc(N * sizeof(float));
	// hipHostAlloc((void**)&x, N * sizeof(float), hipHostMallocDefault);
	// hipHostAlloc((void**)&y, N * sizeof(float), hipHostMallocDefault);

	hipStream_t stream1;
	hipStreamCreate(&stream1);
	hipMalloc(&d_x, N * sizeof(float)); 
	hipMalloc(&d_y, N * sizeof(float));

	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	_lock.unlock();
	// sleep(1);
	cout << "func2 unlocks" << endl;

	double time_1, time_2, latency;
	time_1 = time_now();
	hipError_t e;
	e = hipHostRegister(x, N * sizeof(float), hipHostRegisterDefault);
	cout << hipGetErrorString(e) << endl;
	e = hipHostRegister(y, N * sizeof(float), hipHostRegisterDefault);
	cout << hipGetErrorString(e) << endl;
    time_2 = time_now();
    latency = time_2 - time_1;
	cout << "Latency for pin: " << fixed << latency << endl;

	time_1 = time_now();
    hipMemcpyAsync(d_x, x, N * sizeof(float), hipMemcpyHostToDevice, stream1);
	hipMemcpyAsync(d_y, y, N * sizeof(float), hipMemcpyHostToDevice, stream1);
    time_2 = time_now();
    latency = time_2 - time_1;
	cout << "Latency for tranmission: " << fixed << latency << endl;

	sleep(1);
	hipFree(d_x);
	hipFree(d_y);
	free(x);
	free(y);
	// hipHostFree(x);
	// hipHostFree(y);
}

int main(void) {
	_lock.lock();
	cout << "Locked" << endl;
	sleep(1);

	// Start thread 1
	thread t1(func1);
	sleep(1);

	// Start thread 2
	thread t2(func2);
	sleep(1);

	// Join
	t1.join();
	t2.join();

	return 0;
}