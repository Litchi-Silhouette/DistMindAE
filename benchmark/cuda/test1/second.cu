#include "hip/hip_runtime.h"
#include <unistd.h>

#include <iostream>
#include <string>

#include "../../source/utils/common/time.h"
#include "../../source/utils/tcp/tcp.h"

using namespace std;
using namespace balance::util;

__global__
void saxpy(int n, float a, float *x, float *y)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n) y[i] = a * x[i] + y[i];
}

int main(void) {
	TcpClient client(string("127.0.0.1"), 7777);

	int N = 1 << 25;
	float *x, *y, *d_x, *d_y;
	x = (float*)malloc(N * sizeof(float));
	y = (float*)malloc(N * sizeof(float));

	hipMalloc(&d_x, N * sizeof(float)); 
	hipMalloc(&d_y, N * sizeof(float));

	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

	char buffer[8];
	memset(buffer, 0, 8);
	client.tcpRecv(buffer, 4);
	memcpy(buffer, "WXYZ", 4);
	client.tcpSend(buffer, 4);
	usleep(2000);

    double time_1 = time_now();
    for (int i = 0; i < 1000; ++i) {
        saxpy<<<(N + 255) / 256, 256>>>(N, 2.0f, d_x, d_y);
    }
    // hipDeviceSynchronize();
    double time_2 = time_now();
    double latency = time_2 - time_1;
    cout << "Latency: " << fixed << latency << endl;

	hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

	float maxError = 0.0f;
	for (int i = 0; i < N; i++)
		maxError = max(maxError, abs(y[i] - 2.0f * 1001));
	cout << "Max error: " << fixed << maxError << endl;

	hipFree(d_x);
	hipFree(d_y);
	free(x);
	free(y);
}