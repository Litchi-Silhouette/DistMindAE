#include <unistd.h>
#include <string.h>

#include <iostream>
#include <string>

#include "../../source/utils/common/time.h"
#include "../../source/utils/tcp/tcp.h"

using namespace std;
using namespace balance::util;

int main(void) {
	TcpServer server(string("0.0.0.0"), 7777);
	TcpAgent agent = server.tcpAccept();

	int N = 1 << 25;
	float *x, *y, *d_x, *d_y;
	x = (float*)malloc(N * sizeof(float));
	y = (float*)malloc(N * sizeof(float));
	// hipHostAlloc((void**)&x, N * sizeof(float), hipHostMallocDefault);
	// hipHostAlloc((void**)&y, N * sizeof(float), hipHostMallocDefault);

	hipStream_t stream1;
	hipStreamCreate(&stream1);
	hipMalloc(&d_x, N * sizeof(float)); 
	hipMalloc(&d_y, N * sizeof(float));

	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	char buffer[8];
	memset(buffer, 0, 8);
	memcpy(buffer, "ABCD", 4);
	agent.tcpSend(buffer, 4);
	agent.tcpRecv(buffer, 4);
	cout << buffer << endl;

	double time_1, time_2, latency;
	time_1 = time_now();
	hipError_t e;
	e = hipHostRegister(x, N * sizeof(float), hipHostRegisterDefault);
	cout << hipGetErrorString(e) << endl;
	e = hipHostRegister(y, N * sizeof(float), hipHostRegisterDefault);
	cout << hipGetErrorString(e) << endl;
    time_2 = time_now();
    latency = time_2 - time_1;
	cout << "Latency: " << fixed << latency << endl;

	time_1 = time_now();
    hipMemcpyAsync(d_x, x, N * sizeof(float), hipMemcpyHostToDevice, stream1);
	hipMemcpyAsync(d_y, y, N * sizeof(float), hipMemcpyHostToDevice, stream1);
    time_2 = time_now();
    latency = time_2 - time_1;
	cout << "Latency: " << fixed << latency << endl;

	sleep(1);
	hipFree(d_x);
	hipFree(d_y);
	free(x);
	free(y);
	// hipHostFree(x);
	// hipHostFree(y);
}