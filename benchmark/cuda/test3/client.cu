#include <unistd.h>
#include <stdlib.h>
#include <string.h>

#include <iostream>
#include <string>

#include "../../../source/utils/shared_memory/shared_memory.h"
#include "../../../source/utils/tcp/tcp.h"
#include "../../../source/utils/common/time.h"

using namespace std;
using namespace balance::util;

int main(int argc, char** argv) {
    // Connect to the server
    TcpClient client(string("127.0.0.1"), 7777);
    char buffer[8];

    // Create shared memory
    string name("test-pin");
    size_t size = 256 * 1024 * 1024;
    SharedMemory shm(name, size, true);
    memset(shm.getPointer(), 0, shm.getSize());
    int *data = (int*)shm.getPointer();
    int n = shm.getSize() / sizeof(int);

    // Create cuda stream
    hipStream_t stream1;
    hipStreamCreate(&stream1);
    int *d_data;
    hipMalloc(&d_data, shm.getSize()); 

    // Signal Reader
    memset(buffer, 0, 8);
    memcpy(buffer, "ABCD", 4);
    client.tcpSend(buffer, 4);

    // Get signal
    memset(buffer, 0, 8);
    client.tcpRecv(buffer, 4);
    cout << buffer << endl;

    // Read something
    double time_1, time_2, latency;
    hipError_t e;

	time_1 = time_now();
	e = hipHostRegister(shm.getPointer(), shm.getSize(), hipHostRegisterDefault);
    time_2 = time_now();
    cout << hipGetErrorString(e) << endl;
    latency = time_2 - time_1;
    cout << "Latency for pinning memory: " << fixed << latency << endl;
    
    time_1 = time_now();
    e = hipMemcpyAsync(d_data, data, shm.getSize(), hipMemcpyHostToDevice, stream1);
    time_2 = time_now();
    cout << hipGetErrorString(e) << endl;
    latency = time_2 - time_1;
    cout << "Latency for transmission: " << fixed << latency << endl;

    return 0;
}