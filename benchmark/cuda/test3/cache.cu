#include <unistd.h>
#include <stdlib.h>
#include <string.h>

#include <iostream>
#include <string>

#include "../../../source/utils/shared_memory/shared_memory.h"
#include "../../../source/utils/tcp/tcp.h"
#include "../../../source/utils/common/time.h"

using namespace std;
using namespace balance::util;

int main(int argc, char** argv) {
    // Open shared memory
    string name("test-pin");
    size_t size = 256 * 1024 * 1024;
    SharedMemory shm(name, size, true);
    memset(shm.getPointer(), 0, shm.getSize());
    int *data = (int*)shm.getPointer();
    int n = shm.getSize() / sizeof(int);
    sleep(1);

    // Write something
    for (int i = 0; i < n; ++i)
        data[i] = i;

    // Pin memory
    double time_1, time_2, latency;
    hipError_t e;
    time_1 = time_now();
	e = hipHostRegister(shm.getPointer(), shm.getSize(), hipHostRegisterDefault);
    time_2 = time_now();
    cout << hipGetErrorString(e) << endl;
    latency = time_2 - time_1;
    cout << "Latency for pinning memory: " << fixed << latency << endl;

    // Accept connection
    TcpServer server(string("127.0.0.1"), 7777);
    TcpAgent* agent = server.tcpAccept();
    char buffer[8];

    // Get signal
    memset(buffer, 0, 8);
    agent->tcpRecv(buffer, 4);
    cout << buffer << endl;

    // Signal writer
    memset(buffer, 0, 8);
    memcpy(buffer, "WXYZ", 4);
    agent->tcpSend(buffer, 4);

    // Wait for completion
    sleep(1);

    delete agent;

    return 0;
}